#include "hip/hip_runtime.h"
﻿#pragma once

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cmath>

#include "../Coursework/cuda.cuh"

//constants
__device__ __constant__ double gravConst = 6.6743e-11; //gravitational constant
__device__ __constant__ double sunMass = 1.98847e+31;
__device__ __constant__ double AU = 1.496e+11;
__device__ __constant__ double timeStep = 60 * 60 * 24; //amount of time between each frame
__device__ __constant__ double pi = 3.141592653589793238462643;
__device__ __constant__ double minDistance = 5e+3; //minimum distance between two asteroids to calculate force
__device__ __constant__ double minDistanceSquared = 2.5e+7;


__global__ void StartVelocity(double* xPos, double* yPos, double* zPos, double* xVel, double* yVel, double* zVel, unsigned int numberOfAsteroids)
{
    //assign a thread on a block to an index number of the arrays
    int i = threadIdx.x + (blockDim.x * blockIdx.x);

    if (i < numberOfAsteroids)
    {
        /*
        direction vector from asteroid to sun

        direction = sunPosition - asteroidPosition
        sun is at center 0,0,0
        */
        double xDirAsteroidToSun = 0 - xPos[i];
        double yDirAsteroidToSun = 0 - yPos[i];
        double zDirAsteroidToSun = 0 - zPos[i];

        /*
        direction vector of asteroid

        rotating vectors
        x2 = cos(a) * x1 - sin(a) * y1
        y2 = sin(a) * x1 - cos(a) * y1

        rotated by 90 degrees to find tangent of circle
        x2 = cos(90) * x1 - sin(90) * y1
        y2 = sin(90) * x1 - cos(90) * y1

        simplifies to
        x2 = -y1
        y2 = x1
        */
        double xDirOfAsteroid = -yDirAsteroidToSun;
        double yDirOfAsteroid = xDirAsteroidToSun;
        double zDirOfAsteroid = zDirAsteroidToSun;


        //distance of asteroid from sun = sqrt((x2 - x1)^2 + (y2 - y1)^2 + (z2 - z1)^2)
        double distance = sqrt((xDirAsteroidToSun * xDirAsteroidToSun) + (yDirAsteroidToSun * yDirAsteroidToSun) + (zDirAsteroidToSun * zDirAsteroidToSun));
        //v = sqrt(GM / r)
        //v = inital speed, G = gravitational constant, M = mass of sun, r = distance between asteroid and sun
        double speed = sqrt((gravConst * sunMass) / distance);
        //magnitude of direction vector of asteroid = sqrt(x^2 + y^2 + z^2)
        double magnitude = sqrt(((xDirOfAsteroid) * (xDirOfAsteroid)) + ((yDirOfAsteroid) * (yDirOfAsteroid)) + ((zDirOfAsteroid) * (zDirOfAsteroid)));

        //initial velocity of asteroid = (speed / magnitude) * direction of asteroid vector
        xVel[i] = (speed / magnitude) * xDirOfAsteroid;
        yVel[i] = (speed / magnitude) * yDirOfAsteroid;
        zVel[i] = (speed / magnitude) * zDirOfAsteroid;

        //new position = old position + (velocity * time)
        //s = vt
        //s = position, v = velocity, t = time
        xPos[i] += xVel[i] * timeStep;
        yPos[i] += yVel[i] * timeStep;
        zPos[i] += zVel[i] * timeStep;
    }

}

__global__ void CalculateForce(double* xPos, double* yPos, double* zPos, double* xVel, double* yVel, double* zVel, double* mass, unsigned int numberOfAsteroids)
{
    //assign a thread on a block to an index number of the arrays
    int i = threadIdx.x + (blockDim.x * blockIdx.x);

    double xForce = 0;
    double yForce = 0;
    double zForce = 0;

    double xDistance;
    double yDistance;
    double zDistance;
    double distanceSqaured;
    double distance;


    if (i < numberOfAsteroids)
    {
        for (int j = 0; j < numberOfAsteroids; j++)
        {
            if (i != j)
            {
                //calculate the distance between two asteroids on each axis
                //distance = pos2- pos1
                xDistance = xPos[j] - xPos[i];
                yDistance = yPos[j] - yPos[i];
                zDistance = zPos[j] - zPos[i];
                //distance between asteroids = sqrt((x2 - x1)^2 + (y2 - y1)^2 + (z2 - z1)^2)
                distanceSqaured = (xDistance * xDistance) + (yDistance * yDistance) + (zDistance * zDistance);
                distance = sqrt(distanceSqaured);


                //check if the two asteroids are less than 0.05AU  apart
                if (distanceSqaured < ((0.05 * AU) * (0.05 * AU)))
                {
                    /*
                    calculate force between asteroids
                    F = (G * M1 * M2) / r^2 - scalaer force
                    F-> = (G * M1 * M2 * r->) / r^3 - vector force

                    F-> = force vector, G = gravitational constant, M1 & M2 = masses of the two asteroids, r-> = vector distance, r = scalar distance
                    */

                    //check if distance is less than minimum distance
                    if (distanceSqaured < minDistanceSquared)
                    {
                        //use the minimum distance for the calculation
                        xForce += (gravConst * mass[i] * mass[j] * minDistance) / (minDistance * minDistance * minDistance);
                        yForce += (gravConst * mass[i] * mass[j] * minDistance) / (minDistance * minDistance * minDistance);
                        zForce += (gravConst * mass[i] * mass[j] * minDistance) / (minDistance * minDistance * minDistance);
                    }
                    else
                    {
                        //use the distance for the calculation
                        xForce += (gravConst * mass[i] * mass[j] * xDistance) / (distance * distance * distance);
                        yForce += (gravConst * mass[i] * mass[j] * yDistance) / (distance * distance * distance);
                        zForce += (gravConst * mass[i] * mass[j] * zDistance) / (distance * distance * distance);
                    }

                }


            }

        }

        //calculate the distance between asteroid and sun on each axis
        //distance = pos2- pos1
        //sun is at center 0,0,0
        xDistance = 0 - xPos[i];
        yDistance = 0 - yPos[i];
        zDistance = 0 - zPos[i];
        //distance between asteroids = sqrt((x2 - x1)^2 + (y2 - y1)^2 + (z2 - z1)^2)
        distance = sqrt((xDistance * xDistance) + (yDistance * yDistance) + (zDistance * zDistance));


        //check for zero division
        if (distance > 0)
        {
            /*
            calculate force between asteroid and sun
            F = (G * M1 * M2) / r^2 - scalaer force
            F-> = (G * M1 * M2 * r->) / r^3 - vector force

            F-> = force vector, G = gravitational constant, M1 & M2 = masses of the asteroid and sun, r-> = vector distance, r = scalar distance
            */
            xForce += (gravConst * mass[i] * sunMass * xDistance) / (distance * distance * distance);
            yForce += (gravConst * mass[i] * sunMass * yDistance) / (distance * distance * distance);
            zForce += (gravConst * mass[i] * sunMass * zDistance) / (distance * distance * distance);
        }


        /*
        calculate new velocityand position of asteroid
        a = F/m
        v = at

        a = acceleration, F = force, m = mass, v = velocity, t = time
        */
        xVel[i] += (xForce / mass[i]) * timeStep;
        yVel[i] += (yForce / mass[i]) * timeStep;
        zVel[i] = (zForce / mass[i]) * timeStep;

        //s = vt
        //s = position, v = velocity, t = time
        xPos[i] += xVel[i] * timeStep;
        yPos[i] += yVel[i] * timeStep;
        zPos[i] += zVel[i] * timeStep;


        xForce = 0;
        yForce = 0;
        zForce = 0;


        //printf("%f , %f, %f \n", xForce1, yForce1, zForce1);
        //printf("%f, %f, %f \n", xPos[0], yPos[0], zPos[0]);
    }

}



hipError_t CalculatePosition(double* xPos, double* yPos, double* zPos, double* xVel, double* yVel, double* zVel, double* mass, unsigned int numberOfAsteroids)
{
    double *dev_xPos = 0;
    double *dev_yPos = 0;
    double *dev_zPos = 0;
    double *dev_xVel = 0;
    double *dev_yVel = 0;
    double *dev_zVel = 0;
    double *dev_mass = 0;
    hipError_t cudaStatus;


    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }



    // Allocate GPU buffers
    cudaStatus = hipMalloc((void**)&dev_xPos, numberOfAsteroids * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_yPos, numberOfAsteroids * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_zPos, numberOfAsteroids * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_xVel, numberOfAsteroids * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_yVel, numberOfAsteroids * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_zVel, numberOfAsteroids * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_mass, numberOfAsteroids * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }



    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_xPos, xPos, numberOfAsteroids * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_yPos, yPos, numberOfAsteroids * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_zPos, zPos, numberOfAsteroids * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_xVel, xVel, numberOfAsteroids * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_yVel, yVel, numberOfAsteroids * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_zVel, zVel, numberOfAsteroids * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_mass, mass, numberOfAsteroids * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


    int numberOfBlocks = 1;
    if (numberOfAsteroids > 512)
        numberOfBlocks = (numberOfAsteroids / 512) + 1;
    // Launch a kernel on the GPU with one thread for each element.
    CalculateForce << < numberOfBlocks, 512 >> > (dev_xPos, dev_yPos, dev_zPos, dev_xVel, dev_yVel, dev_zVel, dev_mass, numberOfAsteroids);
    // Check for any errors launching the kernel 
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }


    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(xPos, dev_xPos, numberOfAsteroids * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(yPos, dev_yPos, numberOfAsteroids * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(zPos, dev_zPos, numberOfAsteroids * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(xVel, dev_xVel, numberOfAsteroids * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(yVel, dev_yVel, numberOfAsteroids * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(zVel, dev_zVel, numberOfAsteroids * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(mass, dev_mass, numberOfAsteroids * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


Error:
    hipFree(dev_xPos);
    hipFree(dev_yPos);
    hipFree(dev_zPos);
    hipFree(dev_xVel);
    hipFree(dev_yVel);
    hipFree(dev_zVel);
    hipFree(dev_mass);

    return cudaStatus;
}


hipError_t CalculateStartVelocity(double* xPos, double* yPos, double* zPos, double* xVel, double* yVel, double* zVel, unsigned int numberOfAsteroids)
{
    double* dev_xPos = 0;
    double* dev_yPos = 0;
    double* dev_zPos = 0;
    double* dev_xVel = 0;
    double* dev_yVel = 0;
    double* dev_zVel = 0;
    hipError_t cudaStatus;


    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }



    // Allocate GPU buffers
    cudaStatus = hipMalloc((void**)&dev_xPos, numberOfAsteroids * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_yPos, numberOfAsteroids * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_zPos, numberOfAsteroids * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_xVel, numberOfAsteroids * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_yVel, numberOfAsteroids * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_zVel, numberOfAsteroids * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }



    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_xPos, xPos, numberOfAsteroids * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_yPos, yPos, numberOfAsteroids * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_zPos, zPos, numberOfAsteroids * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_xVel, xVel, numberOfAsteroids * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_yVel, yVel, numberOfAsteroids * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_zVel, zVel, numberOfAsteroids * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }



    int numberOfBlocks = 1;
    if (numberOfAsteroids > 512)
        numberOfBlocks = (numberOfAsteroids / 512) + 1;
    //Launch a kernel on the GPU with one thread for each element.
    StartVelocity << < numberOfBlocks, 512 >> > (dev_xPos, dev_yPos, dev_zPos, dev_xVel, dev_yVel, dev_zVel, numberOfAsteroids);
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }


    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(xPos, dev_xPos, numberOfAsteroids * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(yPos, dev_yPos, numberOfAsteroids * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(zPos, dev_zPos, numberOfAsteroids * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(xVel, dev_xVel, numberOfAsteroids * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(yVel, dev_yVel, numberOfAsteroids * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(zVel, dev_zVel, numberOfAsteroids * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }



Error:
    hipFree(dev_xPos);
    hipFree(dev_yPos);
    hipFree(dev_zPos);
    hipFree(dev_xVel);
    hipFree(dev_yVel);
    hipFree(dev_zVel);

    return cudaStatus;
}
